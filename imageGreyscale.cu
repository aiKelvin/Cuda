#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  for(int r = 0; r < numRows; ++r){
    for(int c = 0; c < numCols; ++c){
        uchar4 rgba = rgbaImage[r * numCols + c];
        float channelSum = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
        greyImage[r * numCols + c] = channelSum;
    }
}
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  int numDevices, ThreadsPerBlock;
  //Stored the number of devices on GPU side
  hipDeviceProp_t prop;
  hipGetDeviceCount(&numDevices);
  
  /*Looped through each device and determined the minmum number of threads per block
  assuming both devices would be used. */

  for(int i = 0; i < numDevices; i++){
    hipGetDeviceProperties(&prop, i);
    if (prop.maxThreadsPerBlock < ThreadsPerBlock){
     ThreadsPerBlock = prop.maxThreadsPerBlock;
    }
  }
  printf("Num devices: %d\n", numDevices);
  printf("Min Threads per block: %d\n", ThreadsPerBlock);
  
  //Called the kernel function 
  const dim3 blockSize(ThreadsPerBlock);
  const dim3 gridSize(numRows * numCols / ThreadsPerBlock);
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
